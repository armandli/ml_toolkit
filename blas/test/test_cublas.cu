
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>

#include <cassert>
#include <ctime>
#include <random>
#include <iostream>

#define gpu_errchk(ans) { gpu_assert((ans), __FILE__, __LINE__); }
inline void gpu_assert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {   
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }   
}

#define BSZ 128
#define TSZ 16
#define SZ (BSZ * TSZ)

using namespace std;

struct Mtx {
  double* data;
  size_t rows;
  size_t cols;
  bool is_cuda;

  Mtx(bool is_cuda, size_t rows, size_t cols):
    data(nullptr), rows(rows), cols(cols), is_cuda(is_cuda) {
    if (is_cuda) { gpu_errchk(hipMalloc(&data, sizeof(double) * rows * cols)); }
    else         data = new double[rows * cols];
  }
  ~Mtx(){
    if (is_cuda) { gpu_errchk(hipFree(data)); }
    else         delete[] data;
  }
};

ostream& operator<<(ostream& out, Mtx& m){
  assert(m.is_cuda == false);

  for (size_t i = 0; i < m.rows; ++i){
    for (size_t j = 0; j < m.cols; ++j)
      cout << m.data[i * m.cols + j] << " ";
    cout << endl;
  }
  return out;
}

//only can generate random floats
//void random_matrix_cuda(Mtx& m){
//  assert(m.is_cuda);
//
//  curandGenerator_t prng;
//  curandCreateGenerator(&prng, CURAND_RNG_PSEUDO_DEFAULT);
//
//  curandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
//  curandGenerateUniform(prng, m.data, m.rows * m.cols);
//}

default_random_engine& get_default_random_engine(){
  static default_random_engine eng(time(0));
  return eng;
}

void random_matrix(Mtx& m){
  assert(m.is_cuda == false);

  uniform_real_distribution<double> dist(-100.F, 100.F);
  default_random_engine& eng = get_default_random_engine();

  for (size_t i = 0; i < m.rows * m.cols; ++i)
    m.data[i] = dist(eng);
}

void generate_sample(Mtx& a, Mtx& b){
  assert(a.is_cuda == false && b.is_cuda == false);
  for (size_t i = 0; i < a.rows * a.cols; ++i)
    a.data[i] = (double)i;
  for (size_t i = 0; i < b.rows * b.cols; ++i)
    b.data[i] = (double)i;
}

clock_t matrix_multiply(Mtx& c, Mtx& a, Mtx& b){
  for (size_t i = 0; i < c.rows; ++i)
    for (size_t j = 0; j < c.cols; ++j){
      c.data[i * c.cols + j] = 0.;
      for (size_t k = 0; k < a.cols; ++k)
        c.data[i * c.cols + j] += a.data[i * a.cols + k] * b.data[k * b.cols + j];
    }

  return clock();
}

void mmul_cublas(Mtx& c, Mtx& a, Mtx& b){
  int lda = a.rows, ldb = a.cols, ldc = a.rows;
  const double alpha = 1.;
  const double beta = 0.;
  const double* palpha = &alpha;
  const double* pbeta = &beta;

  //recommended to save the handle for multiple use
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, a.rows, b.cols, a.cols, palpha, a.data, lda, b.data, ldb, pbeta, c.data, ldc);

  hipblasDestroy(handle);
}

int main(){
  Mtx a(false, SZ, SZ), b(false, SZ, SZ), c(false, SZ, SZ), d(false, SZ, SZ);
  Mtx da(true, SZ, SZ), db(true, SZ, SZ), dc(true, SZ, SZ);

  generate_sample(a, b);

//  random_matrix(a);
//  random_matrix(b);

  clock_t timing_start = clock();
  hipMemcpy(da.data, a.data, sizeof(double) * a.rows * a.cols, hipMemcpyHostToDevice);
  hipMemcpy(db.data, b.data, sizeof(double) * b.rows * b.cols, hipMemcpyHostToDevice);

  mmul_cublas(dc, da, db);

  hipMemcpy(c.data, dc.data, sizeof(double) * c.rows * c.cols, hipMemcpyDeviceToHost);
  cout << "Time: " << (clock() - timing_start) / (double)(CLOCKS_PER_SEC / 1000) << " ms" << endl;

  timing_start = clock();
  clock_t timing_end = matrix_multiply(d, a, b);

  bool is_same = true;
  for (size_t i = 0; i < c.rows; ++i)
    for (size_t j = 0; j < c.cols; ++j)
      if (c.data[i] != d.data[i]){
        cout << "Result Unequal" << endl;
        is_same = false;
        break;
      }
  if (is_same) cout << "Result equal" << endl;
}
